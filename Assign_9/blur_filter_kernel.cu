#include "hip/hip_runtime.h"
/* Blur filter. Device code. */

#ifndef _BLUR_FILTER_KERNEL_H_
#define _BLUR_FILTER_KERNEL_H_

#include "blur_filter.h"

__global__ void 
blur_filter_kernel (const float *in, float *out, int size)
{
    /* Obtain index of thread within the overall execution grid */
    int thread_id = blockIdx.x * blockDim.x + threadIdx.x; 
    /* Compute the stride length = total number of threads */
    int stride = blockDim.x * gridDim.x; 		  

    int pix, i, j;
    int row, col;
    int curr_row, curr_col;
    float blur_value;
    int num_neighbors;


    while (thread_id < (size*size)) {

	pix = thread_id;
        row = pix/size;             /* Obtain row number of pixel */
        col = pix % size;           /* Obtain column number of pixel */

        /* Apply blur filter to current pixel */
        blur_value = 0.0;
        num_neighbors = 0;
        for (i = -BLUR_SIZE; i < (BLUR_SIZE + 1); i++) {
            for (j = -BLUR_SIZE; j < (BLUR_SIZE + 1); j++) {
                /* Accumulate values of neighbors while checking for 
                 * boundary conditions */
                curr_row = row + i;
                curr_col = col + j;
                if ((curr_row > -1) && (curr_row < size) &&\
                        (curr_col > -1) && (curr_col < size)) {
                    blur_value += in[curr_row * size + curr_col];
                    num_neighbors += 1;
                }
            }
        }

        /* Write averaged blurred value out */
        out[pix] = blur_value/num_neighbors;
 
        thread_id += stride;
    }
    return;
}

#endif /* _BLUR_FILTER_KERNEL_H_ */
